#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "common/cpu_bitmap.h"
#define DIM 1080
#define MAX_ITER 500
struct hipComplex{
    float r;
    float i;
    __device__ hipComplex(float a ,float b):r(a),i(b){}
    __device__ float magnitude2(void) {return r*r+i*i;}
    __device__ hipComplex operator*(const hipComplex& a){
        return hipComplex(r*a.r-i*a.i,i*a.r+r*a.i);
    }
    __device__ hipComplex operator+(const hipComplex& a){
        return hipComplex(r+a.r,i+a.i);
    }
};

__device__ int IfJulia(int x,int y){
    const float scale=2;
    float jx=scale * (float)(DIM/2-x)/(DIM/2);
    float jy=scale * (float)(DIM/2-y)/(DIM/2);

    hipComplex c(-0.8,0.156);
    hipComplex a(jx,jy);
    for(int i=0;i<MAX_ITER;i++){
        a=a*a+c;
        if(a.magnitude2()>2){
            return 0;
        }
    }
    return 1;
}

__global__ void render(unsigned char* bitmap){
    int x=blockIdx.x;
    int y=blockIdx.y;
    int tid=x+y*gridDim.x;

    int value=IfJulia(x,y);

    bitmap[tid*4+0]=value*255;
    bitmap[tid*4+1]=0;
    bitmap[tid*4+2]=0;
    bitmap[tid*4+3]=255;

}

int main(){

    CPUBitmap bitmap(DIM,DIM);
    unsigned char *ptr = bitmap.get_ptr();
    unsigned char *dev_bitmap;
    // test the bit map
    // for(int i=0;i<DIM/2;i++){
    //     for(int j=0;j<DIM;j++){
    //         ptr[(i*DIM+j)*4]=255;
    //         ptr[(i*DIM+j)*4+1]=0;
    //         ptr[(i*DIM+j)*4+2]=0;
    //         ptr[(i*DIM+j)*4+3]=255;
    //     }
    // }

    // init the resource for cuda
    hipMalloc((void**)&dev_bitmap,bitmap.image_size());
    dim3 grid(DIM,DIM);

    render<<< grid,1 >>>(dev_bitmap);

    hipMemcpy(bitmap.get_ptr(),dev_bitmap,bitmap.image_size(),hipMemcpyDeviceToHost);

    

    bitmap.display_and_exit();
    hipFree(dev_bitmap);

}